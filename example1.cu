
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void calculate(int *a, int *b, int *c){
    c[threadIdx.x] = ((a[threadIdx.x]+2)+b[threadIdx.x])*3;   
}

#define N 512
int main(void){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    //Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    //Setup input values
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    //Copy input from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    //Launch one add() kernel block on GPU with N threads
    calculate<<<1,N>>>(d_a, d_b, d_c);

    //Copy result back to host
    hipMemcpy(d_c, c, size, hipMemcpyDeviceToHost);

    //Display the result
    printf("The result is %d",*c);

    //Cleanup
    free(a);free(b);free(c);
    hipFree(d_a);hipFree(d_b);hipFree(d_c);

    //Exit program
    return 0;
}